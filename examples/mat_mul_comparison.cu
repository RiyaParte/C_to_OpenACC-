 

#include <hip/hip_runtime.h>
#include<stdio.h> 
#include<time.h>
#include<stdlib.h>
#define THREADS_PER_BLOCK 1024

__global__ void func1(int *b,int *c,int *a,int sum,int k,int col,int row,int n,int m)
{
int i = blockIdx.y*blockDim.y + threadIdx.y;
int j = blockIdx.x*blockDim.x + threadIdx.x;
// printf("i = %d j = %d \n",i,j );
if(i >= 0 && i < m)
{
	if(j >= 0 && j < n) 
		{ 
		  a[(i * n) + j] = i + j;
		} 
	}
}	
__global__ void func2(int *b,int *c,int *a,int sum,int k,int col,int row,int n)
{
int i = blockIdx.y*blockDim.y + threadIdx.y;
int j = blockIdx.x*blockDim.x + threadIdx.x;
	if(i >= 0 && i < n)
	{
		if(j >= 0 && j < k)

		{ 
		  b[(i * k) + j] = i + j; 
		}
	}
}	
__global__ void func3(int *b,int *c,int *a,int i,int sum,int k,int j,int n,int m)
{
int row = blockIdx.y*blockDim.y + threadIdx.y;
int col = blockIdx.x*blockDim.x + threadIdx.x;
if(col >= 0 && col < m)
{
if(row >= 0 && row < k)
{
{
  sum = 0;
  for (i = 0; i < n; i++)
  {
    sum += a[(row * n) + i] * b[(i * k) + col];
  }

  c[(row * k) + col] = sum;
}
}
}
}
int main() {
  int m=800 , n=800 , k=800, i, j, col, row, sum=0;
  int a[m][n], b[n][k], c[m][k];
int *d_b;
int *d_c;
int *d_a; 
int width = n; 
int sqrtThreads = sqrt(THREADS_PER_BLOCK);
int nBlocks = width/sqrtThreads;
if (width % sqrtThreads != 0) { // Add an extra block if necessary
  nBlocks++;
}
dim3 grid(nBlocks, nBlocks, 1);
dim3 block(sqrtThreads, sqrtThreads, 1); // Max number of threads per block

float timespentGPU,timespentGPU1,timespentGPU2,timespentGPU3;
float timespentCPU,timespentCPU1,timespentCPU2,timespentCPU3;

hipEvent_t start, stop; 
hipEventCreate(&start); //Creates an event object 
hipEventCreate(&stop);  
hipMalloc((void **)&d_b, n*k*sizeof(int));
hipMemcpy(d_b, &b, n*k*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_c, m*k*sizeof(int));
hipMemcpy(d_c, &c, m*k*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_a, m*n*sizeof(int));
hipMemcpy(d_a, &a, m*n*sizeof(int), hipMemcpyHostToDevice);
hipEventRecord(start, 0); 
func1<<<grid, block>>>(d_b,d_c,d_a,sum,k,col,row,n,m);
hipEventRecord(stop, 0);
hipEventSynchronize(stop); 
hipEventElapsedTime(&timespentGPU1, start, stop); 
hipDeviceSynchronize();
 hipMemcpy(&b, d_b, n*k*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_b);
hipMemcpy(&c, d_c, m*k*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_c);
hipMemcpy(&a, d_a, m*n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_a);
hipMalloc((void **)&d_b, n*k*sizeof(int));
hipMemcpy(d_b, &b, n*k*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_c, m*k*sizeof(int));
hipMemcpy(d_c, &c, m*k*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_a, m*n*sizeof(int));
hipMemcpy(d_a, &a, m*n*sizeof(int), hipMemcpyHostToDevice); 
 
hipEventRecord(start, 0); 
func2<<<grid, block>>>(d_b,d_c,d_a,sum,k,col,row,n);
hipEventRecord(stop, 0);
hipEventSynchronize(stop); 
hipEventElapsedTime(&timespentGPU2, start, stop); 
hipDeviceSynchronize();
hipDeviceSynchronize();
 hipMemcpy(&b, d_b, n*k*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_b);
hipMemcpy(&c, d_c, m*k*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_c);
hipMemcpy(&a, d_a, m*n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_a);
hipMalloc((void **)&d_b, n*k*sizeof(int));
hipMemcpy(d_b, &b, n*k*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_c, m*k*sizeof(int));
hipMemcpy(d_c, &c, m*k*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_a, m*n*sizeof(int));
hipMemcpy(d_a, &a, m*n*sizeof(int), hipMemcpyHostToDevice); 
hipEventRecord(start, 0); 
func3<<<grid, block>>>(d_b,d_c,d_a,i,sum,k,j,n,m);
hipEventRecord(stop, 0);
hipEventSynchronize(stop); 
hipEventElapsedTime(&timespentGPU3, start, stop); 
hipDeviceSynchronize();
 hipMemcpy(&b, d_b, n*k*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_b);
hipMemcpy(&c, d_c, m*k*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_c);
hipMemcpy(&a, d_a, m*n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_a);
  for (i = 0; i < m; ++i) {
    for (j = 0; j < k; ++j) {
      printf("%d ", c[i][j]);
    }
    printf("\n");
  }  


timespentGPU = timespentGPU1+timespentGPU2+timespentGPU3; 
printf("\n timespent on GPU=%f",timespentGPU);
 


int *h_a=&a[0][0],*h_b=&b[0][0],*h_c=&c[0][0];

hipEventRecord(start, 0); 
  for (i = 0; i < m; ++i) {
    for (j = 0; j < n; ++j) {
      h_a[i * n + j] = i + j;
    }
  }   
hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
hipEventElapsedTime(&timespentCPU1, start, stop); 


hipEventRecord(start, 0); 
  for (i = 0; i < n; ++i) {
    for (j = 0; j < k; ++j) {
      h_b[i * k + j] = i + j;
    }
  } 
hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
hipEventElapsedTime(&timespentCPU2, start, stop);
   
hipEventRecord(start, 0);    
  for (col = 0; col < m; ++col) {
      for (row = 0; row < k; ++row) { 
          sum = 0; 
        for (i = 0; i < n; i++)
        {
          sum += h_a[row * n + i] * h_b[i * k + col];
        }
        h_c[row * k + col] = sum;
      }
  } 
hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
hipEventElapsedTime(&timespentCPU3, start, stop);
timespentCPU = timespentCPU1+timespentCPU2+timespentCPU3;
printf("\n timespent on CPU=%f",timespentCPU);
 

printf("\n Speedup = %f",timespentCPU/timespentGPU) ;
  return 0;
}