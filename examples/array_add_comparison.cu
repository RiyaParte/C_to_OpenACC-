
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include<time.h>
#include<stdlib.h> 
#include<math.h>  
  
__global__ void func1(int *c,int *a,int *b, int n)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
// printf("i = %d\n", i);
if(i < n)
{
a[i] = 2;
b[i] = 3;

}
}
__global__ void func2(int *c,int *a,int *b, int n)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
if(i < n)
{ 
c[i] = a[i] + b[i];

}
}
int main()
{
float timespentGPU,timespentGPU1,timespentGPU2;
float timespentCPU,timespentCPU1,timespentCPU2;


hipEvent_t start, stop; 
hipEventCreate(&start); //Creates an event object 
hipEventCreate(&stop);

hipEventRecord(start, 0);
int *d_c;
int *d_a;
int *d_b;
int n=2*1000000 ;
// int a[n],b[n],c[n];  
int *h_a=(int*) malloc(n* sizeof(int));
int *h_b=(int*) malloc(n * sizeof(int));
int *h_c=(int*) malloc(n * sizeof(int));  
int i ;     
int blocks = 1024;
int threads= 1024;
// printf("Here");
hipMalloc((void **)&d_c, n*sizeof(int));
hipMemcpy(d_c, h_c, n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_a, n*sizeof(int));
hipMemcpy(d_a, h_a, n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_b, n*sizeof(int));
hipMemcpy(d_b, h_b, n*sizeof(int), hipMemcpyHostToDevice); 

hipEventRecord(start, 0);
func1<<<blocks, threads>>>(d_c,d_a,d_b,n);
hipEventRecord(stop, 0);
hipEventSynchronize(stop); 
hipEventElapsedTime(&timespentGPU1, start, stop); 
hipDeviceSynchronize();
 hipMemcpy(h_c, d_c, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_c);
hipMemcpy(h_a, d_a, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_a);
hipMemcpy(h_b, d_b, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_b);
hipMalloc((void **)&d_c, n*sizeof(int));
hipMemcpy(d_c, h_c, n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_a, n*sizeof(int));
hipMemcpy(d_a, h_a, n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_b, n*sizeof(int));
hipMemcpy(d_b, h_b, n*sizeof(int), hipMemcpyHostToDevice); 
hipEventRecord(start, 0); 
func2<<<blocks, threads>>>(d_c,d_a,d_b, n);
hipEventRecord(stop, 0);
hipEventSynchronize(stop); 
hipEventElapsedTime(&timespentGPU2, start, stop); 

hipDeviceSynchronize();
 hipMemcpy(h_c, d_c, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_c);
hipMemcpy(h_a, d_a, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_a);
hipMemcpy(h_b, d_b, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_b);
for (i=0;i<n;i++)
{
 printf("c =%d\n",h_c[i]);
}
timespentGPU = timespentGPU1+timespentGPU2;
printf("\n timespent on GPU=%f ms",timespentGPU);

 	   
hipEventRecord(start, 0); 
for(i = 0;i<n;i++)
{
    h_a[i] =  2;
    h_b[i] =  3;
}  
hipEventRecord(stop, 0);
hipEventSynchronize(stop); 
hipEventElapsedTime(&timespentCPU1, start, stop); 


hipEventRecord(start, 0); 

for (i=0;i<n;i++)
{
    h_c[i] = h_a[i] + h_b[i];
} 
hipEventRecord(stop, 0);
hipEventSynchronize(stop); 
hipEventElapsedTime(&timespentCPU2, start, stop); 


timespentCPU = timespentCPU1+timespentCPU2;
printf("\n timespent on CPU=%f ms",timespentCPU);


printf("\n Speedup = %f",timespentCPU/timespentGPU);
  return 0;
}
