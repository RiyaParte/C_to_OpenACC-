
#include <hip/hip_runtime.h>
#include<stdio.h> 
#include<time.h>
#include<stdlib.h> 
__global__ void func1(int *c,int *a,int *b,int n,int startvalue)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
if( i < n && i >= startvalue )
{
a[i] = i * 2;
b[i] = i * 3;
i++;

}
}
__global__ void func2(int *c,int *a,int *b,int n,int startvalue)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
if( i < n && i >= startvalue )
{
c[i] = a[i] + b[i];
i++;

}
}
int main()
{
int *d_c;
int *d_a;
int *d_b;
int n=5,x;
int a[n],b[n],c[n];  
int i ;     
i=0;
int startvalue;
int blocks = 1024;
int threads= 1024;
hipMalloc((void **)&d_c, n*sizeof(int));
hipMemcpy(d_c, &c, n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_a, n*sizeof(int));
hipMemcpy(d_a, &a, n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_b, n*sizeof(int));
hipMemcpy(d_b, &b, n*sizeof(int), hipMemcpyHostToDevice); 
startvalue = i;
func1<<<blocks, threads>>>(d_c,d_a,d_b,n,startvalue);
hipDeviceSynchronize();
 hipMemcpy(&c, d_c, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_c);
hipMemcpy(&a, d_a, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_a);
hipMemcpy(&b, d_b, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_b);
i=0;
hipMalloc((void **)&d_c, n*sizeof(int));
hipMemcpy(d_c, &c, n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_a, n*sizeof(int));
hipMemcpy(d_a, &a, n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_b, n*sizeof(int));
hipMemcpy(d_b, &b, n*sizeof(int), hipMemcpyHostToDevice); 
startvalue = i;
func2<<<blocks, threads>>>(d_c,d_a,d_b,n,startvalue);
hipDeviceSynchronize();
 hipMemcpy(&c, d_c, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_c);
hipMemcpy(&a, d_a, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_a);
hipMemcpy(&b, d_b, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_b);
i=0;
while(i<n){
 printf("c =%d\n",c[i]);
i++;
}
}
