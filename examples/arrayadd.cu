
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include<time.h>
#include<stdlib.h> 
#include<math.h>  
  
__global__ void func1(int *c,int *a,int *b,int n)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
if(i < n)
{
a[i] = 2 * i;
b[i] = 3 * i;

}
}
__global__ void func2(int *c,int *a,int *b,int n)
{
int i = blockIdx.x*blockDim.x + threadIdx.x;
if(i < n)
{
c[i] = a[i] + b[i];

}
}
int main()
{
int *d_c;
int *d_a;
int *d_b;
int n=5;
int a[n],b[n],c[n];  
int i ;     
int blocks = 2048;
int threads= 2048;
hipMalloc((void **)&d_c, n*sizeof(int));
hipMemcpy(d_c, &c, n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_a, n*sizeof(int));
hipMemcpy(d_a, &a, n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_b, n*sizeof(int));
hipMemcpy(d_b, &b, n*sizeof(int), hipMemcpyHostToDevice); 
func1<<<blocks, threads>>>(d_c,d_a,d_b,n);
hipDeviceSynchronize();
 hipMemcpy(&c, d_c, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_c);
hipMemcpy(&a, d_a, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_a);
hipMemcpy(&b, d_b, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_b);
hipMalloc((void **)&d_c, n*sizeof(int));
hipMemcpy(d_c, &c, n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_a, n*sizeof(int));
hipMemcpy(d_a, &a, n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_b, n*sizeof(int));
hipMemcpy(d_b, &b, n*sizeof(int), hipMemcpyHostToDevice); 
func2<<<blocks, threads>>>(d_c,d_a,d_b,n);
hipDeviceSynchronize();
 hipMemcpy(&c, d_c, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_c);
hipMemcpy(&a, d_a, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_a);
hipMemcpy(&b, d_b, n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_b);
for (i=0;i<n;i++)
{
 printf("c =%d\n",c[i]);
}
  return 0;
}
