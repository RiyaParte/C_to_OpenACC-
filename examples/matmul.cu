#include "hip/hip_runtime.h"
#include<stdio.h> 
#include<time.h>
#include<stdlib.h>
__global__ void func1(int *c,int *a,int *b,int col,int n,int row,int k,int sum,int m)
{
int i = blockIdx.y*blockDim.y + threadIdx.y;
int j = blockIdx.x*blockDim.x + threadIdx.x;
if(i < m)
{
if(j < n)
{
  a[(i * n) + j] = i + j;
}

}
}
__global__ void func2(int *c,int *a,int *b,int col,int n,int row,int k,int sum)
{
int i = blockIdx.y*blockDim.y + threadIdx.y;
int j = blockIdx.x*blockDim.x + threadIdx.x;
if(i < n)
{
if(j < k)
{
  b[(i * k) + j] = i + j;
}

}
}
__global__ void func3(int *c,int *a,int *b,int n,int j,int i,int k,int sum,int m)
{
int col = blockIdx.y*blockDim.y + threadIdx.y;
int row = blockIdx.x*blockDim.x + threadIdx.x;
if(col < m)
{
if(row < k)
{
  sum = 0;
  for (i = 0; i < n; i++)
  {
    sum += a[(row * n) + i] * b[(i * k) + col];
  }

  c[(row * k) + col] = sum;
}

}
}
int main() {
  int m=2, n=2, k=2, i, j, col, row, sum=0;
int *d_c;
int *d_a;
int *d_b;
  int a[m][n], b[n][k], c[m][k];
int blocks = 1024;
int threads= 1024;
dim3 threads(blocks, threads);
dim3 blocks(blocks, threads);
hipMalloc((void **)&d_c, m*k*sizeof(int));
hipMemcpy(d_c, &c, m*k*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_a, m*n*sizeof(int));
hipMemcpy(d_a, &a, m*n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_b, n*k*sizeof(int));
hipMemcpy(d_b, &b, n*k*sizeof(int), hipMemcpyHostToDevice); 
func1<<<blocks, threads>>>(d_c,d_a,d_b,col,n,row,k,sum,m);
hipDeviceSynchronize();
 hipMemcpy(&c, d_c, m*k*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_c);
hipMemcpy(&a, d_a, m*n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_a);
hipMemcpy(&b, d_b, n*k*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_b);
hipMalloc((void **)&d_c, m*k*sizeof(int));
hipMemcpy(d_c, &c, m*k*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_a, m*n*sizeof(int));
hipMemcpy(d_a, &a, m*n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_b, n*k*sizeof(int));
hipMemcpy(d_b, &b, n*k*sizeof(int), hipMemcpyHostToDevice); 
func2<<<blocks, threads>>>(d_c,d_a,d_b,col,n,row,k,sum);
hipDeviceSynchronize();
 hipMemcpy(&c, d_c, m*k*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_c);
hipMemcpy(&a, d_a, m*n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_a);
hipMemcpy(&b, d_b, n*k*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_b);
hipMalloc((void **)&d_c, m*k*sizeof(int));
hipMemcpy(d_c, &c, m*k*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_a, m*n*sizeof(int));
hipMemcpy(d_a, &a, m*n*sizeof(int), hipMemcpyHostToDevice); 
hipMalloc((void **)&d_b, n*k*sizeof(int));
hipMemcpy(d_b, &b, n*k*sizeof(int), hipMemcpyHostToDevice); 
func3<<<blocks, threads>>>(d_c,d_a,d_b,n,j,i,k,sum,m);
hipDeviceSynchronize();
 hipMemcpy(&c, d_c, m*k*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_c);
hipMemcpy(&a, d_a, m*n*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_a);
hipMemcpy(&b, d_b, n*k*sizeof(int), hipMemcpyDeviceToHost); 
hipFree(d_b);
  for (i = 0; i < m; ++i) {
    for (j = 0; j < k; ++j) {
      printf("%d ", c[i][j]);
    }
    printf("\n");
  } 

  return 0;
}
